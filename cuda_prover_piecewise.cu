#include <string>
#include <chrono>
#include <memory>
#include <inttypes.h>
#include <numeric>
#include <vector>

#define NDEBUG 1

#include <prover_reference_functions.hpp>
#include "multiexp/reduce.cu"
#include <libff/algebra/curves/mnt753/mnt4753/mnt4753_pp.hpp>
// #include <libff/algebra/curves/mnt753/mnt4753/mnt4753_init.hpp>
// #include <libff/algebra/curves/mnt753/mnt6753/mnt6753_init.hpp>
#include <libff/algebra/curves/mnt753/mnt6753/mnt6753_pp.hpp>
#include <libfqfft/tools/exceptions.hpp>
#include <libfqfft/evaluation_domain/evaluation_domain.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  //hipStreamSynchronize(A);
  // Use ca to store H
  auto H_tmp = ca;

  size_t m = B::domain_get_m(domain);
  // for i in 0 to m: H_tmp[i] *= cb[i]
  B::vector_Fr_muleq(H_tmp, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  m = B::domain_get_m(domain);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  m = B::domain_get_m(domain);
  typename B::vector_Fr *H_res = B::vector_Fr_zeros(m + 1);
  B::vector_Fr_copy_into(H_tmp, H_res, m);
  return H_res;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  fread((void *) &n, sizeof(size_t), 1, input);
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<mnt4753_libsnark> {
    typedef ECp_MNT4 ECp;
    typedef ECp2_MNT4 ECpe;
};

template<>
struct ec_type<mnt6753_libsnark> {
    typedef ECp_MNT6 ECp;
    typedef ECp3_MNT6 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

void *
load_scalars_async_host(size_t n, FILE *inputs)
{
    static constexpr size_t scalar_bytes = ELT_BYTES;
    size_t total_bytes = n * scalar_bytes;
    printf("total scalar bytes host alloc: %zu\n", total_bytes);

    // void *scalars_buffer = (void *) malloc (total_bytes);
    void *scalars_buffer;
    gpuErrchk( hipHostMalloc(&scalars_buffer, total_bytes) );
    if (fread(scalars_buffer, total_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read scalars\n");
        abort();
    }

    return scalars_buffer;
}

template< typename EC >
void *
load_points_affine_host(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;
    printf("total affine bytes: %zu\n", total_aff_bytes);
    // auto mem = allocate_memory(total_aff_bytes, 1);

    // void *aff_bytes_buffer = (void *) malloc (total_aff_bytes);
    void *aff_bytes_buffer;
    gpuErrchk( hipHostMalloc((void **)&aff_bytes_buffer, total_aff_bytes) );
    if (fread(aff_bytes_buffer, total_aff_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read all curve poinst\n");
        abort();
    }
    // printf("aff_bytes_buffer: %d\n", (int *)aff_bytes_buffer + (total_aff_bytes - 96));

    return aff_bytes_buffer;
}

template<typename EC>
size_t
get_aff_total_bytes(size_t n) 
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;
    size_t total_aff_bytes = n * aff_pt_bytes;
    // printf("total affine bytes for %ld points: %zu\n", n, total_aff_bytes);
    return total_aff_bytes;
}

template <typename B>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path,
        const char *preprocessed_path)
{
    B::init_public_params();

    hipFree(0);
    size_t primary_input_size = 1;

    const size_t CHUNKS = 4;

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");
    size_t d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    rewind(params_file);

    printf("d = %zu, m = %zu\n", d, m);

    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    static constexpr int R = 32;
    static constexpr int C = 5;
    
    auto params = B::read_params(params_file, d, m);
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    void *w_host = load_scalars_async_host(m + 1, inputs_file);
    // auto w_ = load_scalars_async(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host2 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host3 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    auto inputs = B::read_input(inputs_file, d, m);
    fclose(inputs_file);
    print_time(t, "load inputs");

    // Used before switching to async memcpy-ing and no unified memory
    // const var *w = w_.get();
    // printf("w: %zu\n", w);

    FILE *preprocessed_file = fopen(preprocessed_path, "r");

    size_t space = ((m + 1) + R - 1) / R;

    print_time(t, "load preprocessing");

    auto t_gpu = t;

    hipStream_t sA, sB1, sB2, sL;

    // hipStream_t sA[CHUNKS], sB1[CHUNKS], sB2[CHUNKS], sL[CHUNKS];

    size_t out_size = space * ECpe::NELTS * ELT_BYTES;
    size_t w_size = (m+1)*ELT_BYTES;
    printf("w_size: %ld\n", w_size);
    printf("out_size: %ld\n", out_size);
    size_t B1_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m + 1));
    size_t B2_mults_size = get_aff_total_bytes<ECpe>(((1U << C) - 1)*(m + 1));
    size_t L_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m - 1));
    printf("B1_mults_size: %ld\n", B1_mults_size);
    printf("B2_mults_size: %ld\n", B2_mults_size);
    printf("L_mults_size: %ld\n", L_mults_size);

    size_t m_chunked = m / CHUNKS;
    printf("m / CHUNKS: %ld\n", m_chunked);
    size_t B_m_chunked = m_chunked;
    size_t L_m_chunked = m_chunked;

    int B_m_chunks[CHUNKS];
    int L_m_chunks[CHUNKS];
    printf("about to allocate out ptrs\n");

    // Previous location for where memory was declared
    // auto A_mults = load_points_affine_async<ECp>(sA, ((1U << C) - 1)*(m + 1), preprocessed_file);
    // auto out_A = allocate_memory(out_size);


    void *B1_mults_host_chunked[CHUNKS];
    void *B2_mults_host_chunked[CHUNKS];
    void *L_mults_host_chunked[CHUNKS];

    // Each of the CHUNK'd arrays is an output of a multiexponentation 
    var_ptr out_B1[CHUNKS];
    var_ptr out_B2[CHUNKS];
    var_ptr out_L[CHUNKS];

    var *host_B1[CHUNKS];
    var *host_B2[CHUNKS];
    var *host_L[CHUNKS];

    printf("about to allocate B1\n");

    // void *B1_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    
    for (size_t chunk = 0; chunk < CHUNKS; chunk++) {
        if (chunk == CHUNKS - 1) {
            B_m_chunks[chunk] = m_chunked + 1;
            L_m_chunks[chunk] = m_chunked - 1;
        } else {
            B_m_chunks[chunk] = m_chunked;
            L_m_chunks[chunk] = m_chunked;
        }
        size_t chunk_offset = get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[0]*chunk);
        size_t chunk_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[chunk]);
        void *source = load_points_affine_host<ECp>(((1U << C) - 1)*B_m_chunks[chunk], preprocessed_file);
        // B1_mults_host_chunked[chunk] = source;
        gpuErrchk( hipHostMalloc((void **)&B1_mults_host_chunked[chunk], chunk_size) );
        std::memcpy(B1_mults_host_chunked[chunk], source, chunk_size);
        printf("chunk, B1_mults_host_chunked[%ld]: %p\n", chunk, B1_mults_host_chunked[chunk]);
        printf("chunk_offset: %ld, chunk_size: %p\n", chunk_offset, chunk_size);

        out_B1[chunk] = allocate_memory(out_size, 1);
        printf("out_B1[%d]: %p\n", chunk, out_B1[chunk].get());

        out_B2[chunk] = allocate_memory(out_size, 1);
        out_L[chunk] = allocate_memory(out_size, 1);

        hipHostMalloc((void **)&host_B1[chunk], out_size);
        // printf("host_B1: %p\n", host_B1[i]);

        hipHostMalloc((void **)&host_B2[chunk], out_size);
        // printf("host_B2: %p\n", host_B2[i]);

        hipHostMalloc((void **)&host_L[chunk], out_size);
    }
    // printf("B1_mults_host: %p\n", B1_mults_host);

    printf("about to allocate B2\n");
    void *B2_mults_host = load_points_affine_host<ECpe>(((1U << C) - 1)*(m + 1), preprocessed_file);

    printf("about to allocate L\n");
    void *L_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m - 1), preprocessed_file);

    fclose(preprocessed_file);

    // printf("about to allocate A\n");
    // ec_reduce_straus<ECp, C, R>(sA, out_A.get(), A_mults.get(), w, m + 1);
    // var *host_A = (var *) malloc (out_size);
    // hipMemcpyAsync((void **)&host_A[0], out_A.get(), out_size, hipMemcpyDeviceToHost, sA);
    
    hipStreamCreateWithFlags(&sB1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sB2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sL, hipStreamNonBlocking);

    // TODO: consider utilizing shared memory for the witness
    // auto w1 = allocate_memory(w_size, 1);
    // auto w2 = allocate_memory(w_size, 1);
    // auto w3 = allocate_memory(w_size, 1);

    size_t w_size_chunked = w_size / CHUNKS;
    printf("w_size / CHUNKS: %ld\n", w_size_chunked);
    size_t B1_mults_size_chunked = B1_mults_size / CHUNKS;
    printf("B1_mults_size / CHUNKS: %ld\n", B1_mults_size_chunked);
    size_t B2_mults_size_chunked = B2_mults_size / CHUNKS;
    printf("B2_mults_size / CHUNKS: %ld\n", B2_mults_size_chunked);
    size_t L_mults_size_chunked = L_mults_size / CHUNKS;
    printf("L_mults_size / CHUNKS: %ld\n", L_mults_size_chunked);

    // TODO: do the same thing with the offsets as the chunks amounts so that we don't have to encapsulate all the CUDA calls in big IF statements
    // Doing this will clean up a lot of the repeated statements seen below
    // size_t w_offset_B_m = 
    printf("ELT_BYTES: %ld\n", ELT_BYTES);

    for (size_t i = 0; i < CHUNKS; i++) {
        if (i == CHUNKS - 1) {
            B_m_chunked = m_chunked + 1;
            // printf("(m + 1) / CHUNKS: %ld\n", B_m_chunked);
            L_m_chunked = m_chunked - 1;
            // printf("(m - 1) / CHUNKS: %ld\n", L_m_chunked);
        }

        auto w1 = allocate_memory(B_m_chunked * ELT_BYTES, 1);
        auto w2 = allocate_memory(B_m_chunked * ELT_BYTES, 1);
        auto w3 = allocate_memory(L_m_chunked * ELT_BYTES, 1);

        printf("w1 and w2 size: %ld\n", B_m_chunked * ELT_BYTES);
        printf("w3 size: %ld\n", L_m_chunked * ELT_BYTES);

        auto B1_mults = allocate_memory(get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunked), 1);
        auto B2_mults = allocate_memory(get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunked), 1);
        auto L_mults = allocate_memory(get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunked), 1);

        // Need to do this because we are working with var * types that index 8 bytes at a time
        // size_t out_size_scaled = (i * out_size) / 8;

        // TODO: check how we chunk through the multiples and witness, possibly have values overlapping when memcpy'ing causing errors

        printf("i * (B_m_chunked) * ELT_BYTES: %p\n", (i * (B_m_chunked) * ELT_BYTES));
        printf("w_host + i * (B_m_chunked) * ELT_BYTES: %p\n", w_host + (i * (B_m_chunked) * ELT_BYTES));
        printf("B_m_chunked * ELT_BYTES: %ld\n", B_m_chunked * ELT_BYTES);
        // hipDeviceSynchronize();

        if (i == CHUNKS - 1) {
            printf("get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]): %ld\n", get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]) );
            // gpuErrchk( 
            //     hipMemcpyAsync(B1_mults.get(), 
            //     B1_mults_host + get_aff_total_bytes<ECp>(((1U << C) - 1)* i * (B_m_chunked - 1)), 
            //     get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunked), 
            //     hipMemcpyHostToDevice, 
            //     sB1) );
            printf("B1_mults_host_chunked[%ld]: %p\n", i, B1_mults_host_chunked[i]);
           
            gpuErrchk( 
                hipMemcpyAsync(B1_mults.get(), 
                B1_mults_host_chunked[i], 
                get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]), 
                hipMemcpyHostToDevice, 
                sB1) );
            printf("B1_mults.get(): %p\n", B1_mults.get());

            gpuErrchk( 
                hipMemcpyAsync(B2_mults.get(), 
                B2_mults_host + get_aff_total_bytes<ECpe>(((1U << C) - 1)* i * (B_m_chunked - 1)), 
                get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunked), 
                hipMemcpyHostToDevice, 
                sB2) );

            gpuErrchk( 
                hipMemcpyAsync(L_mults.get(), 
                L_mults_host + get_aff_total_bytes<ECp>(((1U << C) - 1)* i * (L_m_chunked + 1)), 
                get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunked), 
                hipMemcpyHostToDevice, 
                sL) );
            
            printf("i * (B_m_chunked - 1) * ELT_BYTES: %p\n", (i * (B_m_chunked - 1) * ELT_BYTES));
            printf("w_host + (i * (B_m_chunked - 1) * ELT_BYTES): %p\n", w_host + (i * (B_m_chunked - 1) * ELT_BYTES));
            gpuErrchk( hipMemcpyAsync(w1.get(), ((char *)w_host) + (i * (B_m_chunked - 1) * ELT_BYTES), B_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sB1) ); 
            gpuErrchk( hipMemcpyAsync(w2.get(), ((char *)w_host2)  + (i * (B_m_chunked - 1) * ELT_BYTES), B_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sB2) ); 

            gpuErrchk( hipMemcpyAsync(w3.get(), ((char *)w_host3) + ((2+(i * (L_m_chunked + 1))) * ELT_BYTES), L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) ); 

        } else {
            // gpuErrchk( 
            //     hipMemcpyAsync(B1_mults.get(), 
            //     B1_mults_host + get_aff_total_bytes<ECp>(((1U << C) - 1)* i * B_m_chunked), 
            //     get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunked), 
            //     hipMemcpyHostToDevice, 
            //     sB1) );
            printf("B1_mults_host_chunked[%ld]: %p\n", i, B1_mults_host_chunked[i]);
            printf("get_aff_total_bytes<ECp>(((1U << C) - 1)* i * B_m_chunks[i]): %ld\n", get_aff_total_bytes<ECp>(((1U << C) - 1)* i * B_m_chunks[i]) );
            // printf("B1_mults_host_chunked[%ld]: %p\n", i, B1_mults_host_chunked[get_aff_total_bytes<ECp>(((1U << C) - 1)* i * B_m_chunks[i])]);

            gpuErrchk( 
                hipMemcpyAsync(B1_mults.get(), 
                B1_mults_host_chunked[i], 
                get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]), 
                hipMemcpyHostToDevice, 
                sB1) );
            printf("B1_mults.get(): %p\n", B1_mults.get());

            gpuErrchk( 
                hipMemcpyAsync(B2_mults.get(), 
                B2_mults_host + get_aff_total_bytes<ECpe>(((1U << C) - 1)* i * B_m_chunked), 
                get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunked), 
                hipMemcpyHostToDevice, 
                sB2) );

            gpuErrchk( 
                hipMemcpyAsync(L_mults.get(), 
                L_mults_host + get_aff_total_bytes<ECp>(((1U << C) - 1)* i * L_m_chunked), 
                get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunked), 
                hipMemcpyHostToDevice, 
                sL) );

            gpuErrchk( hipMemcpyAsync(w1.get(), ((char *)w_host) + (i * (B_m_chunked) * ELT_BYTES), B_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sB1) ); 
            gpuErrchk( hipMemcpyAsync(w2.get(), ((char *)w_host2) + (i * (B_m_chunked) * ELT_BYTES), B_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sB2) );

            gpuErrchk( hipMemcpyAsync(w3.get(), ((char *)w_host3) + ((2+(i * L_m_chunked)) * ELT_BYTES), L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) ); 

        }

        hipDeviceSynchronize();

        ec_reduce_straus<ECp, C, R>(sB1, out_B1[i].get(), B1_mults.get(), w1.get(), B_m_chunked);
        // ec_reduce<ECp>(sB1, )
        printf("out of ec reduce B1, on host\n");
        printf("i * B1_mults_size_chunked: %ld\n", i * B1_mults_size_chunked);
        // printf("out_size_scaled: %ld\n", out_size_scaled);

        // gpuErrchk( hipMemcpyAsync(B2_mults.get(), B2_mults_host + get_aff_total_bytes<ECpe>(i * B_m_chunked), get_aff_total_bytes<ECpe>(B_m_chunked), hipMemcpyHostToDevice, sB2) );
        ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2[i].get(), B2_mults.get(), w2.get(), B_m_chunked);
        printf("out of ec reduce B2, on host\n");

        // gpuErrchk( hipMemcpyAsync(L_mults.get(), L_mults_host + (i * L_m_chunked * 2) * ELT_BYTES, 2 * L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) );
        printf("w_host3: %p\n", w_host3);
        printf("w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES: %p\n", w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES);
        // NOTE: it is only + (2 * ELT_LIMBS) as w3 is a var * that jumps by 64 bits. 12 * 64 = 768 bit element
        printf("w3.get(): %p\n", w3.get());
        printf("w3.get() + (primary_input_size + 1) * ELT_LIMBS: %p\n", w3.get() + (primary_input_size + 1) * ELT_LIMBS);
        ec_reduce_straus<ECp, C, R>(sL, out_L[i].get(), L_mults.get(), w3.get(), L_m_chunked);
        printf("out of ec reduce L, on host\n");

        hipDeviceSynchronize();

        printf("out_B1[%d].get(): %p\n", i, out_B1[i].get()); 
        gpuErrchk( hipMemcpyAsync(host_B1[i], out_B1[i].get(), out_size, hipMemcpyDeviceToHost, sB1) );
        printf("initiated B1 copy to host\n");

        printf("out_B2[%d].get(): %p\n", i, out_B2[i].get()); 
        gpuErrchk( hipMemcpyAsync(host_B2[i], out_B2[i].get(), out_size, hipMemcpyDeviceToHost, sB2) );
        printf("initiated B2 copy to host\n");

        gpuErrchk( hipMemcpyAsync(host_L[i], out_L[i].get(), out_size, hipMemcpyDeviceToHost, sL) );
        printf("initiated L copy to host\n");
    }

    print_time(t, "gpu launch");

    G1 *evaluation_At = B::multiexp_G1(B::input_w(inputs), B::params_A(params), m + 1);

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    auto H = B::params_H(params);
    auto coefficients_for_H =
        compute_H<B>(d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));

    G1 *evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);

    print_time(t, "cpu 1");

    // hipDeviceSynchronize();
    //hipStreamSynchronize(sA);
    //G1 *evaluation_At = B::read_pt_ECp(out_A.get());

    hipStreamSynchronize(sB1);
    printf("synchronized sB1\n");
    printf("host_B1: %" PRIu64 "\n", *(host_B1[0]));
    // G1 *evaluation_Bt1 = B::read_pt_ECp(host_B1);
    // G1 *B1_evaluations[CHUNKS];
    std::vector<G1*> B1_evaluations(CHUNKS);

    for (size_t i = 0; i < CHUNKS; i++) {
        B1_evaluations[i] = B::read_pt_ECp(host_B1[i]);
        B::print_G1(B1_evaluations[i]);
    }

    auto binary_op = [](G1 *p1, G1 *p2){return B::G1_add(p1, p2); };
    G1 *final_B1_host = std::accumulate(B1_evaluations.begin() + 1, B1_evaluations.end(), B1_evaluations[0], binary_op);
    printf("final_B1_host:\n");
    B::print_G1(final_B1_host);

    hipStreamSynchronize(sB2);
    printf("synchronized sB2\n");
    printf("host_B2: %" PRIu64 "\n", *(host_B2[0]));
    // G2 *evaluation_Bt2 = B::read_pt_ECpe(host_B2);
    G2 *B2_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        B2_evaluations[i] = B::read_pt_ECpe(host_B2[i]);
    }

    hipStreamSynchronize(sL);
    printf("synchronized sL\n");
    printf("host_L: %" PRIu64 "\n", *(host_L[0]));
    // G1 *evaluation_Lt = B::read_pt_ECp(host_L);
    G1 *L_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        L_evaluations[i] = B::read_pt_ECp(host_L[i]);
    }
    
    G1 *evaluation_Bt1_sum = B1_evaluations[0];
    G2 *evaluation_Bt2 = B2_evaluations[0];
    G1 *evaluation_Lt = L_evaluations[0];
    for (size_t i = 1; i < CHUNKS; i++) {
        printf("before addition:\n");
        B::print_G1(evaluation_Bt1_sum);
        evaluation_Bt1_sum = B::G1_add(evaluation_Bt1_sum, B1_evaluations[i]);
        printf("after addition:\n");
        B::print_G1(evaluation_Bt1_sum);
        evaluation_Bt2 = B::G2_add(evaluation_Bt2, B2_evaluations[i]);
        // B::print_G2(evaluation_Bt2);
        evaluation_Lt = B::G1_add(evaluation_Lt, L_evaluations[i]);
        // B::print_G1(evaluation_Lt);
    }
    G1 *evaluation_Bt1 = evaluation_Bt1_sum;

    print_time(t_gpu, "gpu e2e");

    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), evaluation_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, Lt1_plus_scaled_Bt1);
    
    print_time(t, "cpu 2");

    B::print_G1(evaluation_Bt1);
    B::print_G2(evaluation_Bt2);
    B::print_G1(evaluation_Lt);
    B::print_G1(evaluation_Ht);

    B::groth16_output_write(evaluation_At, evaluation_Bt2, final_C, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    //hipStreamDestroy(sA);
    hipStreamDestroy(sB1);
    hipStreamDestroy(sB2);
    hipStreamDestroy(sL);

    for (size_t chunk = 0; chunk < CHUNKS; chunk++) {
        hipHostFree(B1_mults_host_chunked[chunk]);
        hipHostFree(host_B1[chunk]);
        hipHostFree(host_B2[chunk]);
        hipHostFree(host_L[chunk]);
    }
    // hipHostFree(B1_mults_host_chunked);
    hipHostFree(B2_mults_host);
    hipHostFree(L_mults_host);
    hipHostFree(w_host);
    hipHostFree(w_host2);
    hipHostFree(w_host3);
    // hipHostFree(host_B1);
    // hipHostFree(host_B2);
    // hipHostFree(host_L);

    B::delete_vector_G1(H);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int argc, char **argv) {
  setbuf(stdout, NULL);
  std::string curve(argv[1]);
  std::string mode(argv[2]);

  const char *params_path = argv[3];

  if (mode == "compute") {
      const char *input_path = argv[4];
      const char *output_path = argv[5];

      if (curve == "MNT4753") {
          run_prover<mnt4753_libsnark>(params_path, input_path, output_path, "MNT4753_preprocessed");
      } else if (curve == "MNT6753") {
          // Temporary for testing
          run_prover<mnt6753_libsnark>(params_path, input_path, output_path, "MNT6753_preprocessed");
      }
  } else if (mode == "preprocess") {
#if 0
      if (curve == "MNT4753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      } else if (curve == "MNT6753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      }
#endif
  }

  return 0;
}
