#include <string>
#include <chrono>
#include <inttypes.h>

#define NDEBUG 1

#include <prover_reference_functions.hpp>
#include "multiexp/reduce.cu"
#include <libff/algebra/curves/mnt753/mnt4753/mnt4753_pp.hpp>
// #include <libff/algebra/curves/mnt753/mnt4753/mnt4753_init.hpp>
// #include <libff/algebra/curves/mnt753/mnt6753/mnt6753_init.hpp>
#include <libff/algebra/curves/mnt753/mnt6753/mnt6753_pp.hpp>
#include <libfqfft/tools/exceptions.hpp>
#include <libfqfft/evaluation_domain/evaluation_domain.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  //hipStreamSynchronize(A);
  // Use ca to store H
  auto H_tmp = ca;

  size_t m = B::domain_get_m(domain);
  // for i in 0 to m: H_tmp[i] *= cb[i]
  B::vector_Fr_muleq(H_tmp, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  m = B::domain_get_m(domain);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  m = B::domain_get_m(domain);
  typename B::vector_Fr *H_res = B::vector_Fr_zeros(m + 1);
  B::vector_Fr_copy_into(H_tmp, H_res, m);
  return H_res;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  fread((void *) &n, sizeof(size_t), 1, input);
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<mnt4753_libsnark> {
    typedef ECp_MNT4 ECp;
    typedef ECp2_MNT4 ECpe;
};

template<>
struct ec_type<mnt6753_libsnark> {
    typedef ECp_MNT6 ECp;
    typedef ECp3_MNT6 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

void *
load_scalars_async_host(size_t n, FILE *inputs)
{
    static constexpr size_t scalar_bytes = ELT_BYTES;
    size_t total_bytes = n * scalar_bytes;
    printf("total scalar bytes host alloc: %zu\n", total_bytes);

    // void *scalars_buffer = (void *) malloc (total_bytes);
    void *scalars_buffer;
    hipHostMalloc(&scalars_buffer, total_bytes);
    if (fread(scalars_buffer, total_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read scalars\n");
        abort();
    }

    return scalars_buffer;
}

template< typename EC >
void *
load_points_affine_host(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;
    printf("total affine bytes: %zu\n", total_aff_bytes);
    // auto mem = allocate_memory(total_aff_bytes, 1);

    // void *aff_bytes_buffer = (void *) malloc (total_aff_bytes);
    void *aff_bytes_buffer;
    hipHostMalloc((void **)&aff_bytes_buffer, total_aff_bytes);
    if (fread(aff_bytes_buffer, total_aff_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read all curve poinst\n");
        abort();
    }
    // printf("aff_bytes_buffer: %d\n", (int *)aff_bytes_buffer + (total_aff_bytes - 96));

    return aff_bytes_buffer;
}

template<typename EC>
size_t
get_aff_total_bytes(size_t n) 
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;
    size_t total_aff_bytes = n * aff_pt_bytes;
    // printf("total affine bytes for %ld points: %zu\n", n, total_aff_bytes);
    return total_aff_bytes;
}

template <typename B>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path,
        const char *preprocessed_path)
{
    B::init_public_params();

    hipFree(0);
    size_t primary_input_size = 1;

    const size_t CHUNKS = 4;

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");
    size_t d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    rewind(params_file);

    printf("d = %zu, m = %zu\n", d, m);

    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    static constexpr int R = 32;
    static constexpr int C = 5;
    
    auto params = B::read_params(params_file, d, m);
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    void *w_host = load_scalars_async_host(m + 1, inputs_file);
    // auto w_ = load_scalars_async(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host2 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host3 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    auto inputs = B::read_input(inputs_file, d, m);
    fclose(inputs_file);
    print_time(t, "load inputs");

    // Used before switching to async memcpy-ing and no unified memory
    // const var *w = w_.get();
    // printf("w: %zu\n", w);

    FILE *preprocessed_file = fopen(preprocessed_path, "r");

    size_t space = ((m + 1) + R - 1) / R;

    print_time(t, "load preprocessing");

    auto t_gpu = t;

    hipStream_t sA, sB1, sB2, sL;

    // hipStream_t sA[CHUNKS], sB1[CHUNKS], sB2[CHUNKS], sL[CHUNKS];

    size_t out_size = space * ECpe::NELTS * ELT_BYTES;
    size_t w_size = (m+1)*ELT_BYTES;
    printf("w_size: %ld\n", w_size);
    printf("out_size: %ld\n", out_size);
    size_t B1_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m + 1));
    size_t B2_mults_size = get_aff_total_bytes<ECpe>(((1U << C) - 1)*(m + 1));
    size_t L_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m - 1));
    printf("B1_mults_size: %ld\n", B1_mults_size);
    printf("B1_mults_size: %ld\n", B2_mults_size);
    printf("B1_mults_size: %ld\n", L_mults_size);

    // Previous location for where memory was declared
    // auto A_mults = load_points_affine_async<ECp>(sA, ((1U << C) - 1)*(m + 1), preprocessed_file);
    // auto out_A = allocate_memory(out_size);

    printf("about to allocate B1\n");

    void *B1_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    // printf("B1_mults_host: %p\n", B1_mults_host);

    printf("about to allocate B2\n");
    void *B2_mults_host = load_points_affine_host<ECpe>(((1U << C) - 1)*(m + 1), preprocessed_file);

    printf("about to allocate L\n");
    void *L_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m - 1), preprocessed_file);

    fclose(preprocessed_file);

    size_t out_size_chunked = out_size * CHUNKS;
    // size_t out_size_chunked = out_size / CHUNKS;
    printf("out_size * CHUNKS: %ld\n", out_size_chunked);

    // Each of the CHUNK'd arrays is an output of a multiexponentation 
    var_ptr out_B1[CHUNKS];
    var_ptr out_B2[CHUNKS];
    var_ptr out_L[CHUNKS];

    var *host_B1[CHUNKS];
    var *host_B2[CHUNKS];
    var *host_L[CHUNKS];

    printf("about to allocate out ptrs\n");
    for (size_t i = 0; i < CHUNKS; i++) {
        out_B1[i] = allocate_memory(out_size, 1);
        out_B2[i] = allocate_memory(out_size, 1);
        out_L[i] = allocate_memory(out_size, 1);

        hipHostMalloc((void **)&host_B1[i], out_size);
        // printf("host_B1: %p\n", host_B1[i]);

        hipHostMalloc((void **)&host_B2[i], out_size);
        // printf("host_B2: %p\n", host_B2[i]);

        hipHostMalloc((void **)&host_L[i], out_size);
        // printf("host_L: %p\n", host_L[i]);
    }
    printf("finished allocating out ptrs\n");

    // auto out_B1 = allocate_memory(out_size_chunked, 1);
    // printf("out_size: %ld\n", out_size);
    // printf("out_B1: %p\n", out_B1.get());

    // auto out_B2 = allocate_memory(out_size_chunked, 1);
    // printf("B2_mults_host: %p\n", B2_mults_host);
    // printf("out_B2: %p\n", out_B2.get());

    // auto out_L = allocate_memory(out_size_chunked, 1);
    // printf("L_mults_host: %p\n", L_mults_host);
    // printf("out_L: %p\n", out_L.get());
    
    // var *host_B1 = nullptr;
    // hipHostMalloc((void **)&host_B1, out_size);
    // printf("host_B1: %p\n", host_B1);
    // // printf("host_B1 + i * out_size_chunked: %p\n", host_B1 + 1 * out_size_chunked); 

    // var *host_B2 = nullptr;
    // hipHostMalloc((void **)&host_B2, out_size);
    // printf("host_B2: %p\n", host_B2);

    // var *host_L = nullptr;
    // hipHostMalloc((void **)&host_L, out_size);
    // printf("host_L: %p\n", host_L);

    // printf("about to allocate A\n");
    // ec_reduce_straus<ECp, C, R>(sA, out_A.get(), A_mults.get(), w, m + 1);
    // var *host_A = (var *) malloc (out_size);
    // hipMemcpyAsync((void **)&host_A[0], out_A.get(), out_size, hipMemcpyDeviceToHost, sA);
    
    hipStreamCreateWithFlags(&sB1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sB2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sL, hipStreamNonBlocking);

    // TODO: consider utilizing shared memory for the witness
    // auto w1 = allocate_memory(w_size, 1);
    // auto w2 = allocate_memory(w_size, 1);
    // auto w3 = allocate_memory(w_size, 1);

    // auto B1_mults = allocate_memory(B1_mults_size, 1);
    // auto B2_mults = allocate_memory(B2_mults_size, 1);
    // auto L_mults = allocate_memory(L_mults_size, 1);
    
    size_t w_size_chunked = w_size / CHUNKS;
    printf("w_size / CHUNKS: %ld\n", w_size_chunked);
    size_t B1_mults_size_chunked = B1_mults_size / CHUNKS;
    printf("B1_mults_size / CHUNKS: %ld\n", B1_mults_size_chunked);
    size_t B2_mults_size_chunked = B2_mults_size / CHUNKS;
    printf("B2_mults_size / CHUNKS: %ld\n", B2_mults_size_chunked);
    size_t L_mults_size_chunked = L_mults_size / CHUNKS;
    printf("L_mults_size / CHUNKS: %ld\n", L_mults_size_chunked);

    size_t m_chunked = m / CHUNKS;
    printf("m / CHUNKS: %ld\n", m_chunked);
    size_t B_m_chunked = m_chunked;
    size_t L_m_chunked = m_chunked;

    // auto x = w_host;
    // size_t testi = 0;
    // while (x != nullptr) {
    //     x = x + testi;
    //     testi++;
    // }
    // printf("testi: %ld\n", testi);

    // TODO: do the same thing with the offsets as the chunks amounts so that we don't have to encapsulate all the CUDA calls in big IF statements
    // size_t w_offset_B_m = 
    printf("ELT_BYTES: %ld\n", ELT_BYTES);
    for (size_t i = 0; i < CHUNKS; i++) {
        if (i == CHUNKS - 1) {
            B_m_chunked = m_chunked + 1;
            // printf("(m + 1) / CHUNKS: %ld\n", B_m_chunked);
            L_m_chunked = m_chunked - 1;
            // printf("(m - 1) / CHUNKS: %ld\n", L_m_chunked);
        }
        printf("last w_size_chunked %ld\n", w_size_chunked);
        printf("B_m_chunked: (m + 1) / CHUNKS: %ld\n", B_m_chunked);
        printf("L_m_chunked: (m - 1) / CHUNKS: %ld\n", L_m_chunked);

        // auto w1 = allocate_memory(w_size_chunked, 1);
        // auto w2 = allocate_memory(w_size_chunked, 1);
        // auto w3 = allocate_memory(w_size_chunked, 1);

        auto w1 = allocate_memory(B_m_chunked * ELT_BYTES, 1);
        auto w2 = allocate_memory(B_m_chunked * ELT_BYTES, 1);
        auto w3 = allocate_memory(L_m_chunked * ELT_BYTES, 1);

        printf("w1: %p\n", w1.get());
        printf("w2: %p\n", w2.get());
        printf("w3: %p\n", w3.get());

        auto B1_mults = allocate_memory(B1_mults_size_chunked, 1);
        auto B2_mults = allocate_memory(B2_mults_size_chunked, 1);
        auto L_mults = allocate_memory(L_mults_size_chunked, 1);

        // auto B1_mults = allocate_memory(get_aff_total_bytes<ECp>(B_m_chunked), 1);
        // auto B2_mults = allocate_memory(get_aff_total_bytes<ECpe>(B_m_chunked), 1);
        // auto L_mults = allocate_memory(get_aff_total_bytes<ECp>(L_m_chunked), 1);

        printf("B1_mults: %p\n", B1_mults.get());
        printf("B2_mults: %p\n", B2_mults.get());
        printf("L_mults: %p\n", L_mults.get());

        // Need to do this because we are working with var * types that index 8 bytes at a time
        // size_t out_size_scaled = (i * out_size) / 8;

        // TODO: try switching how we chunk through the multiple and witness, possible have variabels overlapping causing errors
        // static constexpr int AFF_POINT_LIMBS = 2 * EC::field_type::DEGREE * ELT_LIMBS;

        // gpuErrchk( hipMemcpyAsync(B1_mults.get(), B1_mults_host + i * B1_mults_size_chunked, B1_mults_size_chunked, hipMemcpyHostToDevice, sB1) );

        // TODO: try it with get_aff_total_bytes
        // gpuErrchk( hipMemcpyAsync(B1_mults.get(), B1_mults_host + get_aff_total_bytes<ECp>(i * B_m_chunked), get_aff_total_bytes<ECp>(B_m_chunked), hipMemcpyHostToDevice, sB1) );
        printf("w_host: %p\n", w_host);
        // printf("w_host + (i * B_m_chunked - 1) * ELT_BYTES: %p\n", w_host + (i * B_m_chunked - 1) * ELT_BYTES);
        printf("i * (B_m_chunked) * ELT_BYTES: %p\n", (i * (B_m_chunked) * ELT_BYTES));
        printf("w_host + i * (B_m_chunked) * ELT_BYTES: %p\n", w_host + (i * (B_m_chunked) * ELT_BYTES));
        printf("B_m_chunked * ELT_BYTES: %ld\n", B_m_chunked * ELT_BYTES);
        hipDeviceSynchronize();
        if (i == CHUNKS - 1) {
            gpuErrchk( hipMemcpyAsync(B1_mults.get(), B1_mults_host + get_aff_total_bytes<ECp>(i * (B_m_chunked - 1)), get_aff_total_bytes<ECp>(B_m_chunked), hipMemcpyHostToDevice, sB1) );
            gpuErrchk( hipMemcpyAsync(B2_mults.get(), B2_mults_host + get_aff_total_bytes<ECpe>(i * (B_m_chunked - 1)), get_aff_total_bytes<ECpe>(B_m_chunked), hipMemcpyHostToDevice, sB2) );
            
            
            printf("i * (B_m_chunked - 1) * ELT_BYTES: %p\n", (i * (B_m_chunked - 1) * ELT_BYTES));
            printf("w_host + (i * (B_m_chunked - 1) * ELT_BYTES): %p\n", w_host + (i * (B_m_chunked - 1) * ELT_BYTES));
            gpuErrchk( hipMemcpyAsync(w1.get(), w_host + (i * (B_m_chunked - 1) * ELT_BYTES), (B_m_chunked - 1) * ELT_BYTES, hipMemcpyHostToDevice, sB1) ); 
            gpuErrchk( hipMemcpyAsync((void *)w2.get(), w_host2 + (i * (B_m_chunked - 1) * ELT_BYTES), (B_m_chunked - 1) * ELT_BYTES, hipMemcpyHostToDevice, sB2) ); 
        } else {
            gpuErrchk( hipMemcpyAsync(B1_mults.get(), B1_mults_host + get_aff_total_bytes<ECp>(i * B_m_chunked), get_aff_total_bytes<ECp>(B_m_chunked), hipMemcpyHostToDevice, sB1) );
            gpuErrchk( hipMemcpyAsync(B2_mults.get(), B2_mults_host + get_aff_total_bytes<ECpe>(i * B_m_chunked), get_aff_total_bytes<ECpe>(B_m_chunked), hipMemcpyHostToDevice, sB2) );

            gpuErrchk( hipMemcpyAsync(w1.get(), w_host + (i * (B_m_chunked) * ELT_BYTES), B_m_chunked* ELT_BYTES, hipMemcpyHostToDevice, sB1) ); 
            gpuErrchk( hipMemcpyAsync(w2.get(), w_host2 + (i * (B_m_chunked) * ELT_BYTES), B_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sB2) ); 
        }
        ec_reduce_straus<ECp, C, R>(sB1, out_B1[i].get(), B1_mults.get(), w1.get(), B_m_chunked);
        printf("out of ec reduce B1, on host\n");
        printf("i * B1_mults_size_chunked: %ld\n", i * B1_mults_size_chunked);
        // printf("out_size_scaled: %ld\n", out_size_scaled);
        // printf("host_B1 + out_size_scaled: %p\n", host_B1 + out_size_scaled);
        // printf("out_B1.get() + i * out_size_chunked: %p\n", out_B1[0][i].get() + i * out_size_chunked); 
        printf("out_B1[%d].get(): %p\n", i, out_B1[i].get()); 
        gpuErrchk( hipMemcpyAsync(host_B1[i], out_B1[i].get(), out_size, hipMemcpyDeviceToHost, sB1) );
        printf("initiated B1 copy to host\n");

        // gpuErrchk( hipMemcpyAsync(B2_mults.get(), B2_mults_host + get_aff_total_bytes<ECpe>(i * B_m_chunked), get_aff_total_bytes<ECpe>(B_m_chunked), hipMemcpyHostToDevice, sB2) );
        ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2[i].get(), B2_mults.get(), w2.get(), B_m_chunked);
        printf("out of ec reduce B2, on host\n");
        gpuErrchk( hipMemcpyAsync(host_B2[i], out_B2[i].get(), out_size, hipMemcpyDeviceToHost, sB2) );
        printf("initiated B2 copy to host\n");

        // gpuErrchk( hipMemcpyAsync(L_mults.get(), L_mults_host + (i * L_m_chunked * 2) * ELT_BYTES, 2 * L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) );
        gpuErrchk( hipMemcpyAsync(L_mults.get(), L_mults_host + get_aff_total_bytes<ECpe>(i * L_m_chunked), get_aff_total_bytes<ECpe>(L_m_chunked), hipMemcpyHostToDevice, sL) );
        // gpuErrchk( hipMemcpyAsync(w3.get(), w_host3 + i * w_size_chunked, w_size_chunked, hipMemcpyHostToDevice, sL) ); 
        printf("w_host3: %p\n", w_host3);
        printf("w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES: %p\n", w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES);
        gpuErrchk( hipMemcpyAsync(w3.get(), w_host3 + ((2+(i * L_m_chunked)) * ELT_BYTES), L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) ); 
        // NOTE: it is only + (2 * ELT_LIMBS) as w3 is a var * that jumps by 64 bits. 12 * 64 = 768 bit element
        printf("w3.get(): %p\n", w3.get());
        printf("w3.get() + (primary_input_size + 1) * ELT_LIMBS: %p\n", w3.get() + (primary_input_size + 1) * ELT_LIMBS);
        ec_reduce_straus<ECp, C, R>(sL, out_L[i].get(), L_mults.get(), w3.get() + (primary_input_size + 1) * ELT_LIMBS, L_m_chunked);
        printf("out of ec reduce L, on host\n");
        gpuErrchk( hipMemcpyAsync(host_L[i], out_L[i].get(), out_size, hipMemcpyDeviceToHost, sL) );
        printf("initiated L copy to host\n");
    }

    // printf("about to allocate w 1\n");
    // auto w1 = allocate_memory(w_size, 1);
    // auto w2 = allocate_memory(w_size, 1);
    // auto w3 = allocate_memory(w_size, 1);

    // printf("w1: %p\n", w1.get());
    // printf("w2: %p\n", w2.get());
    // printf("w3: %p\n", w3.get());

    // auto B1_mults = allocate_memory(B1_mults_size, 1);
    // auto B2_mults = allocate_memory(B2_mults_size, 1);
    // auto L_mults = allocate_memory(L_mults_size, 1);

    // // printf("B1_mults_host: %d\n", (int *)B1_mults_host + (total_aff_bytes - 96));
    // hipMemcpyAsync(B1_mults.get(), B1_mults_host, B1_mults_size, hipMemcpyHostToDevice, sB1);
    // hipMemcpyAsync(w1.get(), w_host, w_size, hipMemcpyHostToDevice, sB1); 
    // ec_reduce_straus<ECp, C, R>(sB1, out_B1.get(), B1_mults.get(), w1.get(), m + 1);
    // printf("out of ec reduce B1, on host\n");
    // hipMemcpyAsync(host_B1, out_B1.get(), out_size, hipMemcpyDeviceToHost, sB1);
    // printf("initiated B1 copy to host\n");

    // hipMemcpyAsync(B2_mults.get(), B2_mults_host, B2_mults_size, hipMemcpyHostToDevice, sB2);
    // hipMemcpyAsync(w2.get(), w_host2, w_size, hipMemcpyHostToDevice, sB2); 
    // ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2.get(), B2_mults.get(), w2.get(), m + 1);
    // printf("out of ec reduce B2, on host\n");
    // hipMemcpyAsync(host_B2, out_B2.get(), out_size, hipMemcpyDeviceToHost, sB2);
    // printf("initiated B2 copy to host\n");

    // hipMemcpyAsync(L_mults.get(), L_mults_host, L_mults_size, hipMemcpyHostToDevice, sL);
    // hipMemcpyAsync(w3.get(), w_host3, w_size, hipMemcpyHostToDevice, sL); 
    // ec_reduce_straus<ECp, C, R>(sL, out_L.get(), L_mults.get(), w3.get() + (primary_input_size + 1) * ELT_LIMBS, m - 1);
    // printf("out of ec reduce L, on host\n");
    // hipMemcpyAsync(host_L, out_L.get(), out_size, hipMemcpyDeviceToHost, sL);
    // printf("initiated L copy to host\n");

    // ec_reduce_straus<ECp, C, R>(sB1, out_B1.get(), B1_mults.get(), w1.get(), m + 1);
    // printf("out of ec reduce B1, on host\n");

    print_time(t, "gpu launch");

    G1 *evaluation_At = B::multiexp_G1(B::input_w(inputs), B::params_A(params), m + 1);

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    auto H = B::params_H(params);
    auto coefficients_for_H =
        compute_H<B>(d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));

    G1 *evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);

    print_time(t, "cpu 1");

    // hipDeviceSynchronize();
    //hipStreamSynchronize(sA);
    //G1 *evaluation_At = B::read_pt_ECp(out_A.get());

    hipStreamSynchronize(sB1);
    printf("synchronized sB1\n");
    printf("host_B1: %" PRIu64 "\n", *(host_B1[0]));
    // G1 *evaluation_Bt1 = B::read_pt_ECp(host_B1);
    G1 *B1_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        B1_evaluations[i] = B::read_pt_ECp(host_B1[i]);
        B::print_G1(B1_evaluations[i]);
    }

    hipStreamSynchronize(sB2);
    printf("synchronized sB2\n");
    printf("host_B2: %" PRIu64 "\n", *(host_B2[0]));
    // G2 *evaluation_Bt2 = B::read_pt_ECpe(host_B2);
    G2 *B2_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        B2_evaluations[i] = B::read_pt_ECpe(host_B2[i]);
    }

    hipStreamSynchronize(sL);
    printf("synchronized sL\n");
    printf("host_L: %" PRIu64 "\n", *(host_L[0]));
    // G1 *evaluation_Lt = B::read_pt_ECp(host_L);
    G1 *L_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        L_evaluations[i] = B::read_pt_ECp(host_L[i]);
    }
    
    G1 *evaluation_Bt1 = B1_evaluations[0];
    G2 *evaluation_Bt2 = B2_evaluations[0];
    G1 *evaluation_Lt = L_evaluations[0];
    for (size_t i = 1; i < CHUNKS; i++) {
        evaluation_Bt1 = B::G1_add(evaluation_Bt1, B1_evaluations[i]);
        // B::print_G1(evaluation_Bt1);
        evaluation_Bt2 = B::G2_add(evaluation_Bt2, B2_evaluations[i]);
        // B::print_G2(evaluation_Bt2);
        evaluation_Lt = B::G1_add(evaluation_Lt, L_evaluations[i]);
        // B::print_G1(evaluation_Lt);
    }

    print_time(t_gpu, "gpu e2e");

    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), evaluation_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, Lt1_plus_scaled_Bt1);

    print_time(t, "cpu 2");

    B::print_G1(evaluation_Bt1);
    B::print_G2(evaluation_Bt2);
    B::print_G1(evaluation_Lt);
    B::print_G1(evaluation_Ht);

    B::groth16_output_write(evaluation_At, evaluation_Bt2, final_C, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    //hipStreamDestroy(sA);
    hipStreamDestroy(sB1);
    hipStreamDestroy(sB2);
    hipStreamDestroy(sL);

    hipHostFree(B1_mults_host);
    hipHostFree(B2_mults_host);
    hipHostFree(L_mults_host);
    hipHostFree(w_host);
    hipHostFree(w_host2);
    hipHostFree(w_host3);
    hipHostFree(host_B1);
    hipHostFree(host_B2);
    hipHostFree(host_L);

    B::delete_vector_G1(H);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int argc, char **argv) {
  setbuf(stdout, NULL);
  std::string curve(argv[1]);
  std::string mode(argv[2]);

  const char *params_path = argv[3];

  if (mode == "compute") {
      const char *input_path = argv[4];
      const char *output_path = argv[5];

      if (curve == "MNT4753") {
          run_prover<mnt4753_libsnark>(params_path, input_path, output_path, "MNT4753_preprocessed");
      } else if (curve == "MNT6753") {
          // Temporary for testing
          run_prover<mnt6753_libsnark>(params_path, input_path, output_path, "MNT6753_preprocessed");
      }
  } else if (mode == "preprocess") {
#if 0
      if (curve == "MNT4753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      } else if (curve == "MNT6753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      }
#endif
  }

  return 0;
}
