#include <string>
#include <chrono>
#include <memory>
#include <inttypes.h>
#include <numeric>
#include <vector>

#define NDEBUG 1

#include <prover_reference_functions.hpp>
#include "multiexp/reduce.cu"
#include <libff/algebra/curves/mnt753/mnt4753/mnt4753_pp.hpp>
// #include <libff/algebra/curves/mnt753/mnt4753/mnt4753_init.hpp>
// #include <libff/algebra/curves/mnt753/mnt6753/mnt6753_init.hpp>
#include <libff/algebra/curves/mnt753/mnt6753/mnt6753_pp.hpp>
#include <libfqfft/tools/exceptions.hpp>
#include <libfqfft/evaluation_domain/evaluation_domain.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  //hipStreamSynchronize(A);
  // Use ca to store H
  auto H_tmp = ca;

  size_t m = B::domain_get_m(domain);
  // for i in 0 to m: H_tmp[i] *= cb[i]
  B::vector_Fr_muleq(H_tmp, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  m = B::domain_get_m(domain);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  m = B::domain_get_m(domain);
  typename B::vector_Fr *H_res = B::vector_Fr_zeros(m + 1);
  B::vector_Fr_copy_into(H_tmp, H_res, m);
  return H_res;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  fread((void *) &n, sizeof(size_t), 1, input);
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<mnt4753_libsnark> {
    typedef ECp_MNT4 ECp;
    typedef ECp2_MNT4 ECpe;
};

template<>
struct ec_type<mnt6753_libsnark> {
    typedef ECp_MNT6 ECp;
    typedef ECp3_MNT6 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

void *
load_scalars_async_host(size_t n, FILE *inputs)
{
    static constexpr size_t scalar_bytes = ELT_BYTES;
    size_t total_bytes = n * scalar_bytes;
    printf("total scalar bytes host alloc: %zu\n", total_bytes);

    // void *scalars_buffer = (void *) malloc (total_bytes);
    void *scalars_buffer;
    hipHostMalloc(&scalars_buffer, total_bytes);
    if (fread(scalars_buffer, total_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read scalars\n");
        abort();
    }

    return scalars_buffer;
}

template< typename EC >
void *
load_points_affine_host(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;
    printf("total affine bytes: %zu\n", total_aff_bytes);
    // auto mem = allocate_memory(total_aff_bytes, 1);

    // void *aff_bytes_buffer = (void *) malloc (total_aff_bytes);
    void *aff_bytes_buffer;
    hipHostMalloc((void **)&aff_bytes_buffer, total_aff_bytes);
    if (fread(aff_bytes_buffer, total_aff_bytes, 1, inputs) < 1) {
        fprintf(stderr, "Failed to read all curve poinst\n");
        abort();
    }
    // printf("aff_bytes_buffer: %d\n", (int *)aff_bytes_buffer + (total_aff_bytes - 96));

    return aff_bytes_buffer;
}

template<typename EC>
size_t
get_aff_total_bytes(size_t n) 
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;
    size_t total_aff_bytes = n * aff_pt_bytes;
    // printf("total affine bytes for %ld points: %zu\n", n, total_aff_bytes);
    return total_aff_bytes;
}

template <typename B>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path,
        const char *preprocessed_path)
{
    B::init_public_params();

    hipFree(0);
    size_t primary_input_size = 1;

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");
    size_t d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    rewind(params_file);

    printf("d = %zu, m = %zu\n", d, m);

    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    static constexpr int R = 32;
    static constexpr int C = 5;
    
    auto params = B::read_params(params_file, d, m);
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    void *w_host = load_scalars_async_host(m + 1, inputs_file);
    // auto w_ = load_scalars_async(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host2 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    void *w_host3 = load_scalars_async_host(m + 1, inputs_file);
    rewind(inputs_file);
    auto inputs = B::read_input(inputs_file, d, m);
    fclose(inputs_file);
    print_time(t, "load inputs");

    // Used before switching to async memcpy-ing and no unified memory
    // const var *w = w_.get();
    // printf("w: %zu\n", w);

    FILE *preprocessed_file = fopen(preprocessed_path, "r");

    size_t space = ((m + 1) + R - 1) / R;

    print_time(t, "load preprocessing");

    auto t_gpu = t;

    hipStream_t sA, sB1, sB2, sL;

    // hipStream_t sA[CHUNKS], sB1[CHUNKS], sB2[CHUNKS], sL[CHUNKS];

    size_t out_size = space * ECpe::NELTS * ELT_BYTES;
    size_t w_size = (m+1)*ELT_BYTES;
    printf("w_size: %ld\n", w_size);
    printf("out_size: %ld\n", out_size);
    size_t B1_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m + 1));
    size_t B2_mults_size = get_aff_total_bytes<ECpe>(((1U << C) - 1)*(m + 1));
    size_t L_mults_size = get_aff_total_bytes<ECp>(((1U << C) - 1)*(m - 1));
    size_t total_size = B1_mults_size + B2_mults_size + L_mults_size;
    printf("B1_mults_size: %ld\n", B1_mults_size);
    printf("B2_mults_size: %ld\n", B2_mults_size);
    printf("L_mults_size: %ld\n", L_mults_size);
    printf("total size: %ld\n", total_size);

    size_t *free_device;
    size_t *total_device;

    size_t CHUNKS;
    gpuErrchk( hipMemGetInfo(free_device, total_device) ); 

    if (total_size > *free_device) {

        CHUNKS = total_size / (*free_device - 1);
        printf("CHUNKS: %ld\n", CHUNKS);
    }

    // Previous location for where memory was declared
    // auto A_mults = load_points_affine_async<ECp>(sA, ((1U << C) - 1)*(m + 1), preprocessed_file);
    // auto out_A = allocate_memory(out_size);

    printf("about to allocate B1\n");

    void *B1_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    // printf("B1_mults_host: %p\n", B1_mults_host);

    printf("about to allocate B2\n");
    void *B2_mults_host = load_points_affine_host<ECpe>(((1U << C) - 1)*(m + 1), preprocessed_file);

    printf("about to allocate L\n");
    void *L_mults_host = load_points_affine_host<ECp>(((1U << C) - 1)*(m - 1), preprocessed_file);

    fclose(preprocessed_file);

    // Each of the CHUNK'd arrays is an output of a multiexponentation 
    var_ptr out_B1[CHUNKS];
    var_ptr out_B2[CHUNKS];
    var_ptr out_L[CHUNKS];

    var *host_B1[CHUNKS];
    var *host_B2[CHUNKS];
    var *host_L[CHUNKS];

    void *B1_mults_host_chunked[CHUNKS];
    void *B2_mults_host_chunked[CHUNKS];
    void *L_mults_host_chunked[CHUNKS];


    // originally how memory is laid out for multiples
    // If vec = [P0, ..., Pn], then multiples holds an array
    //
    // [    P0, ...,     Pn,
    //     2P0, ...,    2Pn,
    //     3P0, ...,    3Pn,
    //          ...,
    //  2^(C-1) P0, ..., 2^(C-1) Pn]

    size_t m_chunked = m / CHUNKS;
    printf("m / CHUNKS: %ld\n", m_chunked);
    size_t B_m_chunked = m_chunked;
    size_t L_m_chunked = m_chunked;

    int B_m_chunks[CHUNKS];
    int L_m_chunks[CHUNKS];

    for (size_t chunk = 0; chunk < CHUNKS; chunk++) {
        if (chunk == CHUNKS - 1) {
            B_m_chunks[chunk] = m_chunked + 1;
            L_m_chunks[chunk] = m_chunked - 1;
            // j = chunk * (B_m_chunks[chunk] - 1);

        } else {
            B_m_chunks[chunk] = m_chunked;
            L_m_chunks[chunk] = m_chunked;

            // j = chunk * B_m_chunks[chunk];
        }

        hipHostMalloc((void **)&B1_mults_host_chunked[chunk], get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[chunk]));
        hipHostMalloc((void **)&B2_mults_host_chunked[chunk], get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunks[chunk]));
        hipHostMalloc((void **)&L_mults_host_chunked[chunk], get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunks[chunk]));

    }
    printf("about to allocate out ptrs\n");
    // const char *c_mults = reinterpret_cast<const char *>(B1_mults_host);
    for (size_t chunk = 0; chunk < CHUNKS; chunk++) {
        // We must offset by our common slice amount, as any remaining multiples are processed in final chunk
        size_t B_m_column_offset_chunked = chunk * B_m_chunks[0];
        size_t L_m_column_offset_chunked = chunk * L_m_chunks[0];

        printf("(chunk * B_m_chunks[chunk]): %ld\n", (chunk * B_m_chunks[chunk]));
        printf("get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[chunk]): %ld\n", get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[chunk]));

        size_t B1_len = m+1;
        size_t B2_len = m+1;
        size_t L_len = m-1;
        printf("about to organize chunked multiples arrays\n");
        size_t G1_size = get_aff_total_bytes<ECp>(1);
        printf("G1_size: %ld\n", G1_size);
        size_t G2_size = get_aff_total_bytes<ECpe>(1);
        printf("G2_size: %ld\n", G2_size);
        
        // const char *c_mults_chunked = reinterpret_cast<const char *>(B1_mults_host_chunked[chunk]);

        printf("NEW INNER CHUNK LOOP *********************************************** NEW INNER CHUNK LOOP\n");
        for (size_t i = 0; i < (1U << C) - 1; ++i) {
            size_t B_prev_row_offset = (i-1)*B1_len;
            size_t B_curr_row_offset = i*B1_len;
            size_t L_prev_row_offset = (i-1)*L_len;
            size_t L_curr_row_offset = i*L_len;

            size_t B_chunked_row_offset = B_m_chunks[chunk] * i;
            size_t B_column_bound = B_m_column_offset_chunked + B_m_chunks[chunk];
            // printf("(chunk * j): %ld\n", chunk * j );
            // printf("B_m_chunks[chunk]: %ld\n", B_m_chunks[chunk]);
            size_t aff_bytes_row_plus_col_offset = get_aff_total_bytes<ECp>(B_curr_row_offset + B_m_column_offset_chunked);
            printf("NEW LOOP *********************************************** NEW LOOP\n");
            printf("B_m_column_offset_chunked: %ld, B_column_bound: %ld, i: %ld\n", B_m_column_offset_chunked, B_column_bound, i);
            printf("curr_row_offset: %ld, column_offset: %ld\n", B_curr_row_offset, B_m_column_offset_chunked);
            printf("get_aff_total_bytes<ECp>(curr_row_offset + col_offset): %ld\n", aff_bytes_row_plus_col_offset);
            printf("get_aff_total_bytes<ECp>((B_m_chunks[chunk] * i): %ld\n", get_aff_total_bytes<ECp>((B_m_chunks[chunk] * i)));

            // void *res = c_mults_chunked + get_aff_total_bytes<ECp>(B_m_chunks[chunk] * i);
            // void *source = c_mults + get_aff_total_bytes<ECp>(curr_row_offset + j);
            char *res_B1_mults = ((char *)B1_mults_host_chunked[chunk]) + get_aff_total_bytes<ECp>(B_m_chunks[chunk] * i);
            const char *source_B1_mults = ((const char *)B1_mults_host) + get_aff_total_bytes<ECp>(B_curr_row_offset + B_m_column_offset_chunked);

            // char *c_mults_chunked = reinterpret_cast<char *>(res);
            // const char *c_mults = reinterpret_cast<const char *>(source);

            std::memcpy(res_B1_mults, source_B1_mults, get_aff_total_bytes<ECp>(B_m_chunks[chunk]));
            // gpuErrchk( hipMemcpy(res, source, get_aff_total_bytes<ECp>(B_m_chunks[chunk]), hipMemcpyHostToHost) );

            char *res_B2_mults = ((char *)B2_mults_host_chunked[chunk]) + get_aff_total_bytes<ECpe>(B_m_chunks[chunk] * i);
            const char *source_B2_mults = ((const char *)B2_mults_host) + get_aff_total_bytes<ECpe>(B_curr_row_offset + B_m_column_offset_chunked);
            std::memcpy(res_B2_mults, source_B2_mults, get_aff_total_bytes<ECpe>(B_m_chunks[chunk]));


            char *res_L_mults = ((char *)L_mults_host_chunked[chunk]) + get_aff_total_bytes<ECp>(L_m_chunks[chunk] * i);
            const char *source_L_mults = ((const char *)L_mults_host) + get_aff_total_bytes<ECp>(L_curr_row_offset + L_m_column_offset_chunked);
            std::memcpy(res_L_mults, source_L_mults, get_aff_total_bytes<ECp>(L_m_chunks[chunk]));
        }
        printf("done chunking multiples arrays\n");

        out_B1[chunk] = allocate_memory(out_size, 1);
        out_B2[chunk] = allocate_memory(out_size, 1);
        out_L[chunk] = allocate_memory(out_size, 1);

        hipHostMalloc((void **)&host_B1[chunk], out_size);
        // printf("host_B1: %p\n", host_B1[i]);

        hipHostMalloc((void **)&host_B2[chunk], out_size);
        // printf("host_B2: %p\n", host_B2[i]);

        hipHostMalloc((void **)&host_L[chunk], out_size);
        // printf("host_L: %p\n", host_L[i]);
    }
    printf("finished allocating out ptrs\n");
    
    // printf("about to allocate A\n");
    // ec_reduce_straus<ECp, C, R>(sA, out_A.get(), A_mults.get(), w, m + 1);
    // var *host_A = (var *) malloc (out_size);
    // hipMemcpyAsync((void **)&host_A[0], out_A.get(), out_size, hipMemcpyDeviceToHost, sA);
    
    hipStreamCreateWithFlags(&sB1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sB2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&sL, hipStreamNonBlocking);

    // TODO: consider utilizing shared memory for the witness
    // auto w1 = allocate_memory(w_size, 1);
    // auto w2 = allocate_memory(w_size, 1);
    // auto w3 = allocate_memory(w_size, 1);

    // TODO: do the same thing with the offsets as the chunks amounts so that we don't have to encapsulate all the CUDA calls in big IF statements
    // Doing this will clean up a lot of the repeated statements seen below
    // size_t w_offset_B_m = 
    printf("ELT_BYTES: %ld\n", ELT_BYTES);

    for (size_t i = 0; i < CHUNKS; i++) {

        auto w1 = allocate_memory(B_m_chunks[i] * ELT_BYTES, 1);
        auto w2 = allocate_memory(B_m_chunks[i] * ELT_BYTES, 1);
        auto w3 = allocate_memory(L_m_chunks[i] * ELT_BYTES, 1);

        printf("w1 and w2 size: %ld\n", B_m_chunks[i] * ELT_BYTES);
        printf("w3 size: %ld\n", L_m_chunks[i] * ELT_BYTES);

        auto B1_mults = allocate_memory(get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]), 1);
        auto B2_mults = allocate_memory(get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunks[i]), 1);
        auto L_mults = allocate_memory(get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunks[i]), 1);

        printf("B1_mults and B2_mults size: %ld\n", B_m_chunks[i] * ELT_BYTES);
        printf("L_mults size: %ld\n", L_m_chunks[i] * ELT_BYTES);

        // Need to do this because we are working with var * types that index 8 bytes at a time
        // size_t out_size_scaled = (i * out_size) / 8;

        // TODO: check how we chunk through the multiples and witness, possibly have values overlapping when memcpy'ing causing errors

        // TODO: try it with get_aff_total_bytes
        printf("w_host: %p\n", w_host);
        // printf("w_host + (i * B_m_chunked - 1) * ELT_BYTES: %p\n", w_host + (i * B_m_chunked - 1) * ELT_BYTES);
        printf("i * (B_m_chunked) * ELT_BYTES: %p\n", (i * (B_m_chunks[i]) * ELT_BYTES));
        printf("w_host + i * (B_m_chunked) * ELT_BYTES: %p\n", w_host + (i * (B_m_chunks[i]) * ELT_BYTES));
        printf("B_m_chunked * ELT_BYTES: %ld\n", B_m_chunks[i] * ELT_BYTES);
        // hipDeviceSynchronize();
        gpuErrchk( 
            hipMemcpyAsync(B1_mults.get(), 
            B1_mults_host_chunked[i], 
            get_aff_total_bytes<ECp>(((1U << C) - 1)*B_m_chunks[i]), 
            hipMemcpyHostToDevice, 
            sB1) );

        printf("B1_mults.get(): %p\n", B1_mults.get());

        gpuErrchk( 
            hipMemcpyAsync(B2_mults.get(), 
            B2_mults_host_chunked[i], 
            get_aff_total_bytes<ECpe>(((1U << C) - 1)*B_m_chunks[i]), 
            hipMemcpyHostToDevice, 
            sB2) );

        gpuErrchk( 
            hipMemcpyAsync(L_mults.get(), 
                L_mults_host_chunked[i], 
                get_aff_total_bytes<ECp>(((1U << C) - 1)*L_m_chunks[i]), 
                hipMemcpyHostToDevice, 
                sL) );

        printf("B1_mults_host_chunked[i]: %p\n", B1_mults_host_chunked[i]);
        printf("i * (B_m_chunked - 1) * ELT_BYTES: %p\n", (i * (B_m_chunks[0]) * ELT_BYTES));
        printf("w_host + (i * (B_m_chunked - 1) * ELT_BYTES): %p\n", w_host + (i * (B_m_chunks[0]) * ELT_BYTES));

        gpuErrchk( hipMemcpyAsync(w1.get(), ((char *)w_host) + (i * (B_m_chunks[0]) * ELT_BYTES), B_m_chunks[i] * ELT_BYTES, hipMemcpyHostToDevice, sB1) ); 
        gpuErrchk( hipMemcpyAsync(w2.get(), ((char *)w_host2) + (i * (B_m_chunks[0]) * ELT_BYTES), B_m_chunks[i] * ELT_BYTES, hipMemcpyHostToDevice, sB2) );
        gpuErrchk( hipMemcpyAsync(w3.get(), ((char *)w_host3) + ((2+(i * L_m_chunks[0])) * ELT_BYTES), L_m_chunks[i] * ELT_BYTES, hipMemcpyHostToDevice, sL) ); 

        ec_reduce_straus<ECp, C, R>(sB1, out_B1[i].get(), B1_mults.get(), w1.get(), B_m_chunks[i]);
        // ec_reduce<ECp>(sB1, )
        printf("out of ec reduce B1, on host\n");
        // printf("out_size_scaled: %ld\n", out_size_scaled);

        // gpuErrchk( hipMemcpyAsync(B2_mults.get(), B2_mults_host + get_aff_total_bytes<ECpe>(i * B_m_chunked), get_aff_total_bytes<ECpe>(B_m_chunked), hipMemcpyHostToDevice, sB2) );
        ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2[i].get(), B2_mults.get(), w2.get(), B_m_chunks[i]);
        printf("out of ec reduce B2, on host\n");

        // gpuErrchk( hipMemcpyAsync(L_mults.get(), L_mults_host + (i * L_m_chunked * 2) * ELT_BYTES, 2 * L_m_chunked * ELT_BYTES, hipMemcpyHostToDevice, sL) );
        printf("w_host3: %p\n", w_host3);
        printf("w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES: %p\n", w_host3 + (2+(i * L_m_chunked)) * ELT_BYTES);
        // NOTE: it is only + (2 * ELT_LIMBS) as w3 is a var * that jumps by 64 bits. 12 * 64 = 768 bit element
        printf("w3.get(): %p\n", w3.get());
        printf("w3.get() + (primary_input_size + 1) * ELT_LIMBS: %p\n", w3.get() + (primary_input_size + 1) * ELT_LIMBS);
        ec_reduce_straus<ECp, C, R>(sL, out_L[i].get(), L_mults.get(), w3.get(), L_m_chunks[i]);
        printf("out of ec reduce L, on host\n");

        printf("i: %ld, out_B1[%d].get(): %p\n", i, out_B1[i].get()); 
        gpuErrchk( hipMemcpyAsync(host_B1[i], out_B1[i].get(), out_size, hipMemcpyDeviceToHost, sB1) );
        printf("initiated B1 copy to host\n");

        gpuErrchk( hipMemcpyAsync(host_B2[i], out_B2[i].get(), out_size, hipMemcpyDeviceToHost, sB2) );
        printf("initiated B2 copy to host\n");

        gpuErrchk( hipMemcpyAsync(host_L[i], out_L[i].get(), out_size, hipMemcpyDeviceToHost, sL) );
        printf("initiated L copy to host\n");
    }

    print_time(t, "gpu launch");

    G1 *evaluation_At = B::multiexp_G1(B::input_w(inputs), B::params_A(params), m + 1);

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    auto H = B::params_H(params);
    auto coefficients_for_H =
        compute_H<B>(d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));

    G1 *evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);

    print_time(t, "cpu 1");

    // hipDeviceSynchronize();
    //hipStreamSynchronize(sA);
    //G1 *evaluation_At = B::read_pt_ECp(out_A.get());

    hipStreamSynchronize(sB1);
    printf("synchronized sB1\n");
    printf("host_B1: %" PRIu64 "\n", *(host_B1[0]));
    // G1 *evaluation_Bt1 = B::read_pt_ECp(host_B1);
    // G1 *B1_evaluations[CHUNKS];
    std::vector<G1*> B1_evaluations(CHUNKS);

    for (size_t i = 0; i < CHUNKS; i++) {
        B1_evaluations[i] = B::read_pt_ECp(host_B1[i]);
        B::print_G1(B1_evaluations[i]);
    }

    auto binary_op = [](G1 *p1, G1 *p2){return B::G1_add(p1, p2); };
    G1 *final_B1_host = std::accumulate(B1_evaluations.begin() + 1, B1_evaluations.end(), B1_evaluations[0], binary_op);
    printf("final_B1_host:\n");
    B::print_G1(final_B1_host);

    hipStreamSynchronize(sB2);
    printf("synchronized sB2\n");
    printf("host_B2: %" PRIu64 "\n", *(host_B2[0]));
    // G2 *evaluation_Bt2 = B::read_pt_ECpe(host_B2);
    G2 *B2_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        B2_evaluations[i] = B::read_pt_ECpe(host_B2[i]);
    }

    hipStreamSynchronize(sL);
    printf("synchronized sL\n");
    printf("host_L: %" PRIu64 "\n", *(host_L[0]));
    // G1 *evaluation_Lt = B::read_pt_ECp(host_L);
    G1 *L_evaluations[CHUNKS];
    for (size_t i = 0; i < CHUNKS; i++) {
        L_evaluations[i] = B::read_pt_ECp(host_L[i]);
    }
    
    G1 *evaluation_Bt1_sum = B1_evaluations[0];
    G2 *evaluation_Bt2 = B2_evaluations[0];
    G1 *evaluation_Lt = L_evaluations[0];
    for (size_t i = 1; i < CHUNKS; i++) {
        printf("before addition:\n");
        B::print_G1(evaluation_Bt1_sum);
        evaluation_Bt1_sum = B::G1_add(evaluation_Bt1_sum, B1_evaluations[i]);
        printf("after addition:\n");
        B::print_G1(evaluation_Bt1_sum);
        evaluation_Bt2 = B::G2_add(evaluation_Bt2, B2_evaluations[i]);
        // B::print_G2(evaluation_Bt2);
        evaluation_Lt = B::G1_add(evaluation_Lt, L_evaluations[i]);
        // B::print_G1(evaluation_Lt);
    }
    G1 *evaluation_Bt1 = evaluation_Bt1_sum;

    print_time(t_gpu, "gpu e2e");

    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), evaluation_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, Lt1_plus_scaled_Bt1);
    
    print_time(t, "cpu 2");

    B::print_G1(evaluation_Bt1);
    B::print_G2(evaluation_Bt2);
    B::print_G1(evaluation_Lt);
    B::print_G1(evaluation_Ht);

    B::groth16_output_write(evaluation_At, evaluation_Bt2, final_C, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    //hipStreamDestroy(sA);
    hipStreamDestroy(sB1);
    hipStreamDestroy(sB2);
    hipStreamDestroy(sL);

    hipHostFree(B1_mults_host);
    hipHostFree(B2_mults_host);
    hipHostFree(L_mults_host);
    hipHostFree(w_host);
    hipHostFree(w_host2);
    hipHostFree(w_host3);
    hipHostFree(host_B1);
    hipHostFree(host_B2);
    hipHostFree(host_L);

    B::delete_vector_G1(H);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int argc, char **argv) {
  setbuf(stdout, NULL);
  std::string curve(argv[1]);
  std::string mode(argv[2]);

  const char *params_path = argv[3];

  if (mode == "compute") {
      const char *input_path = argv[4];
      const char *output_path = argv[5];

      if (curve == "MNT4753") {
          run_prover<mnt4753_libsnark>(params_path, input_path, output_path, "MNT4753_preprocessed");
      } else if (curve == "MNT6753") {
          // Temporary for testing
          run_prover<mnt6753_libsnark>(params_path, input_path, output_path, "MNT6753_preprocessed");
      }
  } else if (mode == "preprocess") {
#if 0
      if (curve == "MNT4753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      } else if (curve == "MNT6753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      }
#endif
  }

  return 0;
}
